#include <iostream>
#include <hip/hip_runtime.h>


__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main() {
    int N = 1000;
    size_t size = N * sizeof(float);

    // Allocate host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < N; ++i) {
        h_A[i] = static_cast<float>(i);
        h_B[i] = static_cast<float>(i * 2);
    }

    // Allocate device memory
    float* d_A;
    float* d_B;
    float* d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch the kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    std::cout << "CUDA error: " << hipGetErrorString(hipGetLastError()) << std::endl;

    // Copy result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    std::cout << "CUDA error: " << hipGetErrorString(hipGetLastError()) << std::endl;

    // Verify the result
    for (int i = 0; i < N; ++i) {
        if (h_C[i] != h_A[i] + h_B[i]) {
            std::cerr << "Error at index " << i << ": " << h_C[i] << " != " << h_A[i] + h_B[i] << std::endl;
            return -1;
        }
    }

    std::cout << "Vector addition successful!" << std::endl;

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}